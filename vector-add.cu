
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__
void init(int n, float *x, float val){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for(int i = index; i < n; i += stride){
        x[i] = val;
    }
}

__global__
void add(int n, float *x, float *y){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < n; i += stride){
        y[i] = x[i] + y[i];
    }
}

int main(){
    int N = 1 << 20;
    float *x;
    float *y;
    hipMalloc(&x, sizeof(float)*N);
    hipMalloc(&y, sizeof(float)*N);
    
    int threads_per_block = 512;
    int block_num = (N + threads_per_block - 1) / threads_per_block;
    
    init<<<block_num, threads_per_block>>>(N, x, 1.0f);
    init<<<block_num, threads_per_block>>>(N, y, 2.0f);

    add<<<block_num, threads_per_block>>>(N, x, y);
    
    float *x_host = new float[N];
    float *y_host = new float[N];

    hipMemcpy(x_host, x, sizeof(float)*N, hipMemcpyDeviceToHost);
    hipMemcpy(y_host, y, sizeof(float)*N, hipMemcpyDeviceToHost);

    // sanity check
    float max_error = 0.0f;
    for(int i = 0; i < N; i++){
        max_error = fmax(max_error, fabs(y_host[i] - 3.0f));
    }
    
    std::cout << "Max error: " << max_error << std::endl;

    hipFree(x);
    hipFree(y);
    
    delete[] x_host;
    delete[] y_host;

    return 0;
}